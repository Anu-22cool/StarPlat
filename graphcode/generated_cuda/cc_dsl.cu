// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "cc_dsl.h"

void Compute_CC(graph& g,float* CC)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;
  int *h_src;
  int *h_weight;
  int *h_rev_meta;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp = g.indexofNodes[i];
    h_meta[i] = temp;
    temp = g.rev_indexofNodes[i];
    h_rev_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp = g.edgeList[i];
    h_data[i] = temp;
    temp = g.srcList[i];
    h_src[i] = temp;
    temp = edgeLen[i];
    h_weight[i] = temp;
  }


  int* d_meta;
  int* d_data;
  int* d_src;
  int* d_weight;
  int* d_rev_meta;
  bool* d_modified_next;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));
  hipMalloc(&d_src, sizeof(int)*(E));
  hipMalloc(&d_weight, sizeof(int)*(E));
  hipMalloc(&d_rev_meta, sizeof(int)*(V+1));
  hipMalloc(&d_modified_next, sizeof(bool)*(V));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(   d_src,    h_src, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_weight, h_weight, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_rev_meta, h_rev_meta, sizeof(int)*((V+1)), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? 512: V;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params
  float* d_CC;
  hipMalloc(&d_CC, sizeof(float)*(V));


  //BEGIN DSL PARSING 
  initKernel<float> <<<numBlocks,threadsPerBlock>>>(V,d_CC,(float)0);

  // int V = g.num_nodes( ); // asst in .cu

  int src = 0;
  do{
    int* d_dist;
    hipMalloc(&d_dist, sizeof(int)*(V));

    bool* d_modified;
    hipMalloc(&d_modified, sizeof(bool)*(V));

    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_dist,(int)INT_MAX);

    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_modified,(bool)false);

    initIndex<bool><<<1,1>>>(V,d_modified,src,(bool)true); //InitIndexDevice
    initIndex<int><<<1,1>>>(V,d_dist,src,(int)0); //InitIndexDevice
    bool finished = false; // asst in .cu

    // FIXED POINT variables
    //BEGIN FIXED POINT
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    int k=0; // #fixpt-Iterations
    while(!finished) {

      finished = true;
      hipMemcpyToSymbol(HIP_SYMBOL(::finished), &finished, sizeof(bool), 0, hipMemcpyHostToDevice);
      Compute_CC_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_modified,d_dist);
      hipDeviceSynchronize();




      hipMemcpyFromSymbol(&finished, HIP_SYMBOL(::finished), sizeof(bool), 0, hipMemcpyDeviceToHost);
      hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
      initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
      k++;
    } // END FIXED POINT

    float temp = 0; // asst in .cu

    hipMemcpyToSymbol(HIP_SYMBOL(::temp), &temp, sizeof(float), 0, hipMemcpyHostToDevice);
    Compute_CC_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_dist);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&temp, HIP_SYMBOL(::temp), sizeof(float), 0, hipMemcpyDeviceToHost);



    initIndex<float><<<1,1>>>(V,d_CC,src,(float)(1 / temp)); //InitIndexDevice
    src = src + 1;

    //hipFree up!! all propVars in this BLOCK!
    hipFree(d_modified);
    hipFree(d_dist);

  }while(src < V);

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

  hipMemcpy(      CC,     d_CC, sizeof(float)*(V), hipMemcpyDeviceToHost);
} //end FUN

int main(int argc, char *argv[])
{
  char *filename = argv[1];
  graph g(filename);
  g.parseGraph();
  float *CC = (float *)malloc((g.num_nodes()) * sizeof(float));
  Compute_CC(g, CC);

  for (int i = 0 ; i < g.num_nodes(); i++)
  {
    std::cout << i << " " << CC[i] << std::endl;
  }
  std::cout << std::endl;
}