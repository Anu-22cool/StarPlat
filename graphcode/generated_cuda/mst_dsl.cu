// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "mst_dsl.h"

void Boruvka(graph& g)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;
  int *h_src;
  int *h_weight;
  int *h_rev_meta;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp = g.indexofNodes[i];
    h_meta[i] = temp;
    temp = g.rev_indexofNodes[i];
    h_rev_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp = g.edgeList[i];
    h_data[i] = temp;
    temp = g.srcList[i];
    h_src[i] = temp;
    temp = edgeLen[i];
    h_weight[i] = temp;
  }


  int* d_meta;
  int* d_data;
  int* d_src;
  int* d_weight;
  int* d_rev_meta;
  bool* d_modified_next;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));
  hipMalloc(&d_src, sizeof(int)*(E));
  hipMalloc(&d_weight, sizeof(int)*(E));
  hipMalloc(&d_rev_meta, sizeof(int)*(V+1));
  hipMalloc(&d_modified_next, sizeof(bool)*(V));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(   d_src,    h_src, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_weight, h_weight, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_rev_meta, h_rev_meta, sizeof(int)*((V+1)), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? 512: V;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;
  unsigned numBlocks_Edge    = (E+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params

  //BEGIN DSL PARSING 
  int* d_nodeId;
  hipMalloc(&d_nodeId, sizeof(int)*(V));

  int* d_color;
  hipMalloc(&d_color, sizeof(int)*(V));

  int* d_color_next;
  hipMalloc(&d_color_next, sizeof(int)*(V));

  bool* d_isMSTEdge;
  hipMalloc(&d_isMSTEdge, sizeof(bool)*(E));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_nodeId,(int)-1);

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_color,(int)-1);

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_color_next,(int)-1);

  initKernel<bool> <<<numBlocks_Edge,threadsPerBlock>>>(E,d_isMSTEdge,(bool)false);

  Boruvka_kernel_1<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_color_next,d_color,d_nodeId);
  hipDeviceSynchronize();



  hipMemcpy(d_color_next, d_color, sizeof(int)*V, hipMemcpyDeviceToDevice);
  bool* d_modified;
  hipMalloc(&d_modified, sizeof(bool)*(V));

  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_modified,(bool)false);

  bool noNewComp = false; // asst in .cu

  // FIXED POINT variables
  //BEGIN FIXED POINT
  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
  // int k=0; // #fixpt-Iterations
  while(!noNewComp) {

    noNewComp = true;
    hipMemcpyToSymbol(HIP_SYMBOL(::noNewComp), &noNewComp, sizeof(bool), 0, hipMemcpyHostToDevice);
    int* d_minEdge;
    hipMalloc(&d_minEdge, sizeof(int)*(V));

    int* d_minEdge_next;
    hipMalloc(&d_minEdge_next, sizeof(int)*(V));

    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_minEdge,(int)-1);

    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_minEdge_next,(int)-1);

    Boruvka_kernel_2<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_minEdge,d_color);
    hipDeviceSynchronize();



    ; // asst in .cu

    int* d_minEdgeOfComp;
    hipMalloc(&d_minEdgeOfComp, sizeof(int)*(V));

    int* d_minEdgeOfComp_next;
    hipMalloc(&d_minEdgeOfComp_next, sizeof(int)*(V));

    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_minEdgeOfComp,(int)-1);

    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_minEdgeOfComp_next,(int)-1);

    bool finishedMinEdge = false; // asst in .cu

    // FIXED POINT variables
    //BEGIN FIXED POINT
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
   // int k=0; // #fixpt-Iterations
    while(!finishedMinEdge) {

      finishedMinEdge = true;
      hipMemcpyToSymbol(HIP_SYMBOL(::finishedMinEdge), &finishedMinEdge, sizeof(bool), 0, hipMemcpyHostToDevice);
      hipMemcpy(d_minEdgeOfComp_next, d_minEdgeOfComp, sizeof(int)*V, hipMemcpyDeviceToDevice);
      hipMemcpyToSymbol(HIP_SYMBOL(::finishedMinEdge), &finishedMinEdge, sizeof(bool), 0, hipMemcpyHostToDevice);
      Boruvka_kernel_3<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_minEdge,d_minEdgeOfComp,d_color,d_minEdgeOfComp_next);
      hipDeviceSynchronize();
      hipMemcpyFromSymbol(&finishedMinEdge, HIP_SYMBOL(::finishedMinEdge), sizeof(bool), 0, hipMemcpyDeviceToHost);



      ; // asst in .cu

      ; // asst in .cu

      ; // asst in .cu

      ; // asst in .cu

      ; // asst in .cu

      hipMemcpy(d_minEdgeOfComp, d_minEdgeOfComp_next, sizeof(int)*V, hipMemcpyDeviceToDevice);

      hipMemcpyFromSymbol(&finishedMinEdge, HIP_SYMBOL(::finishedMinEdge), sizeof(bool), 0, hipMemcpyDeviceToHost);
      hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
      initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    } // END FIXED POINT

    Boruvka_kernel_4<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_color,d_nodeId,d_minEdgeOfComp,d_minEdgeOfComp_next);
    hipDeviceSynchronize();



    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    hipMemcpy(d_minEdgeOfComp, d_minEdgeOfComp_next, sizeof(int)*V, hipMemcpyDeviceToDevice);
    Boruvka_kernel_5<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_color,d_nodeId,d_minEdgeOfComp,d_isMSTEdge);
    hipDeviceSynchronize();



    hipMemcpy(d_color_next, d_color, sizeof(int)*V, hipMemcpyDeviceToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::noNewComp), &noNewComp, sizeof(bool), 0, hipMemcpyHostToDevice);
    Boruvka_kernel_6<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_color,d_nodeId,d_minEdgeOfComp,d_color_next);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&noNewComp, HIP_SYMBOL(::noNewComp), sizeof(bool), 0, hipMemcpyDeviceToHost);



    ; // asst in .cu

    hipMemcpy(d_color, d_color_next, sizeof(int)*V, hipMemcpyDeviceToDevice);
    bool finished = false; // asst in .cu

    // FIXED POINT variables
    //BEGIN FIXED POINT
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    int k=0; // #fixpt-Iterations
    while(!finished) {

      finished = true;
      hipMemcpyToSymbol(HIP_SYMBOL(::finished), &finished, sizeof(bool), 0, hipMemcpyHostToDevice);
      hipMemcpy(d_color_next, d_color, sizeof(int)*V, hipMemcpyDeviceToDevice);
      hipMemcpyToSymbol(HIP_SYMBOL(::finished), &finished, sizeof(bool), 0, hipMemcpyHostToDevice);
      Boruvka_kernel_7<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_color,d_color_next);
      hipDeviceSynchronize();
      hipMemcpyFromSymbol(&finished, HIP_SYMBOL(::finished), sizeof(bool), 0, hipMemcpyDeviceToHost);



      ; // asst in .cu

      ; // asst in .cu

      hipMemcpy(d_color, d_color_next, sizeof(int)*V, hipMemcpyDeviceToDevice);

      hipMemcpyFromSymbol(&finished, HIP_SYMBOL(::finished), sizeof(bool), 0, hipMemcpyDeviceToHost);
      hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
      initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    } // END FIXED POINT


    //hipFree up!! all propVars in this BLOCK!
    hipFree(d_minEdgeOfComp_next);
    hipFree(d_minEdgeOfComp);
    hipFree(d_minEdge_next);
    hipFree(d_minEdge);

    hipMemcpyFromSymbol(&noNewComp, HIP_SYMBOL(::noNewComp), sizeof(bool), 0, hipMemcpyDeviceToHost);
    hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
  } // END FIXED POINT


  //hipFree up!! all propVars in this BLOCK!
  

  //TIMER STOP
  bool *h_isMSTEdge = (bool *)malloc(E * sizeof(bool));
  hipMemcpy(h_isMSTEdge, d_isMSTEdge, E * sizeof(bool), hipMemcpyDeviceToHost);
  int mst = 0;
  for (int i = 0; i < E; i++)
  {
    if (h_isMSTEdge[i]==true)
      mst += h_weight[i];
  }

  
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);
  printf("mst=%d\n", mst);

  hipFree(d_modified);
  hipFree(d_isMSTEdge);
  hipFree(d_color_next);
  hipFree(d_color);
  hipFree(d_nodeId);
}
int main(int argc, char *argv[])
{
  char *file_name = argv[1];
  graph g(file_name);
  g.parseGraph();
  Boruvka(g);
  return 0;
}
