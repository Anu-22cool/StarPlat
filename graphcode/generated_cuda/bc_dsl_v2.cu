// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "bc_dsl_v2.h"

void Compute_BC(graph& g,float* BC,std::set<int>& sourceSet)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;
  int *h_src;
  int *h_weight;
  int *h_rev_meta;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp = g.indexofNodes[i];
    h_meta[i] = temp;
    temp = g.rev_indexofNodes[i];
    h_rev_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp = g.edgeList[i];
    h_data[i] = temp;
    temp = g.srcList[i];
    h_src[i] = temp;
    temp = edgeLen[i];
    h_weight[i] = temp;
  }


  int* d_meta;
  int* d_data;
  int* d_src;
  int* d_weight;
  int* d_rev_meta;
  bool* d_modified_next;

  cl_mem d_meta = clCreateBuffer(context, CL_MEM_READ_WRITE, (1+V)*sizeof(int), NULL, &status);
  cl_mem d_data = clCreateBuffer(context, CL_MEM_READ_WRITE, (E)*sizeof(int), NULL, &status);
  cl_mem d_src = clCreateBuffer(context, CL_MEM_READ_WRITE, (E)*sizeof(int), NULL, &status);
  cl_mem d_weight = clCreateBuffer(context, CL_MEM_READ_WRITE, (E)*sizeof(int), NULL, &status);
  cl_mem d_rev_meta = clCreateBuffer(context, CL_MEM_READ_WRITE, (V+1)*sizeof(int), NULL, &status);
  cl_mem d_modified_next = clCreateBuffer(context, CL_MEM_READ_WRITE, (V)*sizeof(bool), NULL, &status);

  status = clEnqueueWriteBuffer(command_queue,   d_meta , CL_TRUE, 0, sizeof(int)*V+1,   h_meta, 0, NULL, NULL );
  status = clEnqueueWriteBuffer(command_queue,   d_data , CL_TRUE, 0, sizeof(int)*E,   h_data, 0, NULL, NULL );
  status = clEnqueueWriteBuffer(command_queue,    d_src , CL_TRUE, 0, sizeof(int)*E,    h_src, 0, NULL, NULL );
  status = clEnqueueWriteBuffer(command_queue, d_weight , CL_TRUE, 0, sizeof(int)*E, h_weight, 0, NULL, NULL );
  status = clEnqueueWriteBuffer(command_queue, d_rev_meta , CL_TRUE, 0, sizeof(int)*(V+1), h_rev_meta, 0, NULL, NULL );

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? 512: V;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params
  float* d_BC;
  hipMalloc(&d_BC, sizeof(float)*(V));


  //BEGIN DSL PARSING 
  initKernel<float> <<<numBlocks,threadsPerBlock>>>(V,d_BC,(float)0);

  //FOR SIGNATURE of SET - Assumes set for on .cu only
  std::set<int>::iterator itr;
  for(itr=sourceSet.begin();itr!=sourceSet.end();itr++) 
  {
    int src = *itr;
    double* d_sigma;
    hipMalloc(&d_sigma, sizeof(double)*(V));

    float* d_delta;
    hipMalloc(&d_delta, sizeof(float)*(V));

    initKernel<float> <<<numBlocks,threadsPerBlock>>>(V,d_delta,(float)0);

    initKernel<double> <<<numBlocks,threadsPerBlock>>>(V,d_sigma,(double)0);

    initIndex<double><<<1,1>>>(V,d_sigma,src,(double)1); //InitIndexDevice

    //EXTRA vars for ITBFS AND REVBFS
    bool finished;
    int hops_from_source=0;
    bool* d_finished;       hipMalloc(&d_finished,sizeof(bool) *(1));
    int* d_hops_from_source;hipMalloc(&d_hops_from_source, sizeof(int));  hipMemset(d_hops_from_source,0,sizeof(int));
    int* d_level;           hipMalloc(&d_level,sizeof(int) *(V));

    //EXTRA vars INITIALIZATION
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_level,-1);
    initIndex<int><<<1,1>>>(V,d_level,src, 0);

    // long k =0 ;// For DEBUG
    do {
      finished = true;
      status = clEnqueueWriteBuffer(command_queue, d_finished , CL_TRUE, 0, sizeof(bool)*1, &finished, 0, NULL, NULL );

      //Kernel LAUNCH
      fwd_pass<<<numBlocks,threadsPerBlock>>>(V, d_meta, d_data,d_weight, d_delta, d_sigma, d_level, d_hops_from_source, d_finished,d_BC); ///DONE from varList

      incrementDeviceVar<<<1,1>>>(d_hops_from_source);
      hipDeviceSynchronize(); //MUST - rupesh
      ++hops_from_source; // updating the level to process in the next iteration
      // k++; //DEBUG

      clEnqueueReadBuffer(command_queue, &finished , CL_TRUE, 0, sizeof(bool)*1, d_finished, 0, NULL, NULL );
    }while(!finished);

    hops_from_source--;
    status = clEnqueueWriteBuffer(command_queue, d_hops_from_source , CL_TRUE, 0, sizeof(int)*1, &hops_from_source, 0, NULL, NULL );

    //BACKWARD PASS
    while(hops_from_source > 1) {

      //KERNEL Launch
      back_pass<<<numBlocks,threadsPerBlock>>>(V, d_meta, d_data, d_weight, d_delta, d_sigma, d_level, d_hops_from_source, d_finished
        ,d_BC); ///DONE from varList

      hops_from_source--;
      status = clEnqueueWriteBuffer(command_queue, d_hops_from_source , CL_TRUE, 0, sizeof(int)*1, &hops_from_source, 0, NULL, NULL );
    }
    //accumulate_bc<<<numBlocks,threadsPerBlock>>>(V,d_delta, d_BC, d_level, src);

    //hipFree up!! all propVars in this BLOCK!
    hipFree(d_delta);
    hipFree(d_sigma);

  }

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

  clEnqueueReadBuffer(command_queue, BC , CL_TRUE, 0, sizeof(float)*V, d_BC, 0, NULL, NULL );
} //end FUN
