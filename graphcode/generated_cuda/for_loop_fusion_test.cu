// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "for_loop_fusion_test.h"

void for_loop_fusion(graph& g,std::set<int>& sourceSet)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;
  int *h_src;
  int *h_weight;
  int *h_rev_meta;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp = g.indexofNodes[i];
    h_meta[i] = temp;
    temp = g.rev_indexofNodes[i];
    h_rev_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp = g.edgeList[i];
    h_data[i] = temp;
    temp = g.srcList[i];
    h_src[i] = temp;
    temp = edgeLen[i];
    h_weight[i] = temp;
  }


  int* d_meta;
  int* d_data;
  int* d_src;
  int* d_weight;
  int* d_rev_meta;
  bool* d_modified_next;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));
  hipMalloc(&d_src, sizeof(int)*(E));
  hipMalloc(&d_weight, sizeof(int)*(E));
  hipMalloc(&d_rev_meta, sizeof(int)*(V+1));
  hipMalloc(&d_modified_next, sizeof(bool)*(V));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(   d_src,    h_src, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_weight, h_weight, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_rev_meta, h_rev_meta, sizeof(int)*((V+1)), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? 512: V;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params

  //BEGIN DSL PARSING 
  int* d_count_prop_1;
  hipMalloc(&d_count_prop_1, sizeof(int)*(V));

  int* d_count_prop_2;
  hipMalloc(&d_count_prop_2, sizeof(int)*(V));

  merged_kernel_1<<<numBlocks,threadsPerBlock>>>(V, d_count_prop_1, (int)1, d_count_prop_2, (int)2);
  int count1 = 0; // asst in .cu

  int count2 = 0; // asst in .cu

  hipMemcpyToSymbol(HIP_SYMBOL(::count1), &count1, sizeof(int), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(::count2), &count2, sizeof(int), 0, hipMemcpyHostToDevice);
  for_loop_fusion_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next);
  hipDeviceSynchronize();




  //hipFree up!! all propVars in this BLOCK!
  hipFree(d_count_prop_2);
  hipFree(d_count_prop_1);

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

} //end FUN
