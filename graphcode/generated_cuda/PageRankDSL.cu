#include "PageRankDSL.h"

void Compute_PR(graph& g,float beta,float delta,int maxIter,
  float* pageRank)

{
  // CSR BEGIN
  unsigned V = g.num_nodes();
  unsigned E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;
  int *h_weight;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp = g.indexofNodes[i];
    h_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp = g.edgeList[i];
    h_data[i] = temp;
    temp = edgeLen[i];
    h_weight[i] = temp;
  }


  int* d_meta;
  int* d_data;
  int* d_weight;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));
  hipMalloc(&d_weight, sizeof(int)*(E));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_weight, h_weight, sizeof(int)*(E), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? 512: V;
  unsigned numBlocks    = (numThreads+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);

  //END CSR 

  //DECLAR DEVICE AND HOST vars in params
  double* d_BC; hipMalloc(&d_BC, sizeof(double)*(V)); ///TODO from func

  //BEGIN DSL PARSING 
  float num_nodes = (float)g.num_nodes( );
  initKernel<float> <<<numBlocks,threadsPerBlock>>>(V,d_pageRank,1 / num_nodes);

  int iterCount = 0;
  float diff = 0.0 ;
  do
  diff = 0.000000;
  unsigned int id = threadIdx.x + (blockDim.x * blockIdx.x);
  unsigned int v =id;
  if (id < V)
  {iterCount++;
    while((diff > beta) && (iterCount < maxIter));//TIMER STOP
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("GPU Time: %.6f ms\n", milliseconds);

    hipMemcpy(BC,d_BC , sizeof(double) * (V), hipMemcpyDeviceToHost);
  } //end FUN
