// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "PageRankDSLV2.h"

void Compute_PR(graph& g,float beta,float delta,int maxIter,
  float* pageRank)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

   FILE* fptr = fopen("num_devices.txt","r"); 
  int devicecount;
   if(fptr == NULL){ 
     hipGetDeviceCount(&devicecount); 
     } 
   else{ 
     fscanf(fptr," %d ",&devicecount); 
     fclose(fptr); 
  }
  int* h_vertex_partition;
  int *h_edges;//data
  int *h_weight;
  int *h_offset;//meta
  int* h_src;
  int *h_rev_meta;
  int h_vertex_per_device;

  h_edges = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_offset = (int *)malloc( (V+1)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_vertex_partition = (int*)malloc((devicecount+1)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));
  h_vertex_per_device = V/devicecount;

  for(int i=0; i<= V; i++) {
    h_offset[i] = g.indexofNodes[i];
    h_rev_meta[i] = g.rev_indexofNodes[i];
  }

  int index = 0;
  h_vertex_partition[0]=0;
  h_vertex_partition[devicecount]=V;
  for(int i=1;i<devicecount;i++){
    if(i<=(V%devicecount)){
       index+=(h_vertex_per_device+1);
    }
    else{
       index+=h_vertex_per_device;
    }
    h_vertex_partition[i]=index;
  }
  for(int i=0; i< E; i++){
    h_edges[i]= g.edgeList[i];
    h_src[i]=g.srcList[i];
    h_weight[i] = edgeLen[i];
  }


  int** d_offset;
  int** d_edges;
  int** d_weight;
  int** d_src;
  int** d_rev_meta;
  d_offset = (int**) malloc(devicecount*sizeof(int*));
  d_edges = (int**) malloc(devicecount*sizeof(int*));
  d_weight = (int**) malloc(devicecount*sizeof(int*));
  d_src = (int**) malloc(devicecount*sizeof(int*));
  d_rev_meta = (int**) malloc(devicecount*sizeof(int*));

  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipMalloc(&d_offset[i], (V+1)*sizeof(int) );
    hipMalloc(&d_edges[i], (E)*sizeof(int) );
    hipMalloc(&d_weight[i], (E)*sizeof(int) );
    hipMalloc(&d_src[i], (E)*sizeof(int) );
    hipMalloc(&d_rev_meta[i], (V+1)*sizeof(int) );
  }
  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipMemcpyAsync(d_offset[i], h_offset, (V+1)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpyAsync(d_edges[i], h_edges, (E)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpyAsync(d_weight[i], h_weight, (E)*sizeof(int),hipMemcpyHostToDevice );
    hipMemcpyAsync(d_src[i], h_src, (E)*sizeof(int),hipMemcpyHostToDevice );
    hipMemcpyAsync(d_rev_meta[i], h_rev_meta, (V+1)*sizeof(int),hipMemcpyHostToDevice );
  }
  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  const unsigned threadsPerBlock = 1024;
  unsigned numThreads   = (V < threadsPerBlock)? V: 1024;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;
  unsigned numBlocksKernel    = (V+threadsPerBlock-1)/threadsPerBlock;
  unsigned numBlocks_Edge    = (E+threadsPerBlock-1)/threadsPerBlock;

  if(devicecount>1){
    numBlocksKernel = numBlocksKernel/devicecount+1;
  }


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLARE DEVICE AND HOST vars in params
  float** d_delta;
  d_delta = (float**)malloc(sizeof(float*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_delta[i], sizeof(float));
    initKernel<float> <<<1,1>>>(1,d_delta[i],delta);
  }

  float** h_delta = (float**)malloc(sizeof(float*)*(devicecount+1));
  for(int i=0;i<=devicecount;i++){
    h_delta[i] = (float*)malloc(sizeof(float));
  }
  float* h_pageRank;
  h_pageRank= (float*)malloc(sizeof(float)*(V+1));
  float** d_pageRank;
  d_pageRank = (float**)malloc(sizeof(float*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_pageRank[i], sizeof(float)*(V+1));
  }


  //BEGIN DSL PARSING 
  float num_nodes = (float)g.num_nodes( ); // asst in .cu 
  //fixed_pt_var
  float** h_num_nodes;
  h_num_nodes = (float**)malloc(sizeof(float*)*(devicecount+1));
  for(int i=0;i<=devicecount;i+=1){
    h_num_nodes[i] = (float*)malloc(sizeof(float));
  }

  float** d_num_nodes;
  d_num_nodes = (float**)malloc(sizeof(float*)*devicecount);
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    hipMalloc(&d_num_nodes[i],sizeof(float));
    initKernel<float> <<<1,1>>>(1,d_num_nodes[i],g.num_nodes( ));
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }


  float* h_pageRank_nxt;
  h_pageRank_nxt=(float*)malloc(sizeof(float)*(V+1));
  float** d_pageRank_nxt;
  d_pageRank_nxt = (float**)malloc(sizeof(float*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_pageRank_nxt[i], sizeof(float)*(V+1));
  }

  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<float> <<<numBlocks,threadsPerBlock>>>(V,d_pageRank[i],(float)1 / num_nodes);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_pageRank+h_vertex_partition[i],d_pageRank[i]+h_vertex_partition[i],(h_vertex_partition[i+1]-h_vertex_partition[i])*sizeof(float),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i+=1){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<float> <<<numBlocks,threadsPerBlock>>>(V,d_pageRank_nxt[i],(float)0);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_pageRank_nxt+h_vertex_partition[i],d_pageRank_nxt[i]+h_vertex_partition[i],(h_vertex_partition[i+1]-h_vertex_partition[i])*sizeof(float),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i+=1){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  int iterCount = 0; // asst in .cu 
  //fixed_pt_var

  float diff; // asst in .cu 
  //fixed_pt_var
  float** h_diff;
  h_diff = (float**)malloc(sizeof(float*)*(devicecount+1));
  for(int i=0;i<=devicecount;i+=1){
    h_diff[i] = (float*)malloc(sizeof(float));
  }

  float** d_diff;
  d_diff = (float**)malloc(sizeof(float*)*devicecount);
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    hipMalloc(&d_diff[i],sizeof(float));
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }


  do{
    diff = 0.000000;
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      //printed here

      initKernel<float> <<<1,1>>>(1,d_diff[i],(float)diff);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      Compute_PR_kernel1<<<numBlocksKernel, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_diff[i],d_delta[i],d_num_nodes[i],d_pageRank[i],d_pageRank_nxt[i]);
    }

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    //fixed_pt_var

    //fixed_pt_var

    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_diff[i],d_diff[i],sizeof(float),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    float diff_=0;
    for(int i=0;i<devicecount;i++){
      diff_ += h_diff[i][0];
    } //end of for
    diff=diff_;
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_diff[i],d_diff[i],sizeof(float),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    float diff_=0;
    for(int i=0;i<devicecount;i++){
      diff_ += h_diff[i][0];
    } //end of for
    diff=diff_;
    if(devicecount>1){
      //v v
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_pageRank_nxt+h_vertex_partition[i],d_pageRank_nxt[i]+h_vertex_partition[i],sizeof(float)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(d_pageRank_nxt[i],h_pageRank_nxt,sizeof(float)*(V+1),hipMemcpyHostToDevice);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpy(d_pageRank[i],d_pageRank_nxt[i],sizeof(float)*(V+1),hipMemcpyDeviceToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    iterCount++;
  }while((diff > beta) && (iterCount < maxIter));
  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

  hipMemcpy(pageRank, d_pageRank, sizeof(float)*(V), hipMemcpyDeviceToHost);
} //end FUN
